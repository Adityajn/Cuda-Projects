
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 2
#define GRID_SIZE 2
#define N GRID_SIZE * BLOCK_SIZE


__global__ void MatrixMul(float *A, float *B, float *C, int n)
{
	// Each thread computes a single element of C
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;

	float sum = 0;
	for (int i = 0; i < n; ++i) {
		sum += (A[row*n + i] * B[i*n + col]);
	}
		
	C[row*n + col] = sum;
	printf("\n Block[%d][%d] : Thread[%d][%d] : Product = %.2f\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, sum);
}

int main()
{
	// Perform matrix multiplication C = A*B
	// where A, B and C are NxN matrices
	// Restricted to matrices where N = GRID_SIZE*BLOCGRID_SIZE_SIZE;
	float *hA, *hB, *hC;
	float *dA, *dB, *dC;
	int size = N * N * sizeof(float);

	printf("Executing Matrix Multiplcation\n");
	printf("Matrix size: %d x %d\n", N,N);

	// Allocate memory on the host
	hA = (float *) malloc(size);
	hB = (float *) malloc(size);
	hC = (float *) malloc(size);

	// Initialize matrices on the host
	for (int j = 0; j<N; j++){
		for (int i = 0; i<N; i++){
			hA[j*N + i] = 2;
			hB[j*N + i] = 1;
		}
	}

	printf("Matrix 1:\n");
	for (int j = 0; j<N; j++){
		for (int i = 0; i<N; i++){
			printf("%.2f ", hA[j*N + i]);
		}
		printf("\n");
	}
	
	printf("\nMatrix 2:\n");
	for (int j = 0; j<N; j++){
		for (int i = 0; i<N; i++){
			printf("%.2f ", hB[j*N + i]);
		}
		printf("\n");
	}
	// Allocate memory on the device
	hipMalloc(&dA, size);
	hipMalloc(&dB, size);
	hipMalloc(&dC, size);

	dim3 threadBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid(GRID_SIZE, GRID_SIZE);

	// Copy matrices from the host to device
	hipMemcpy(dA, hA, size, hipMemcpyHostToDevice);
	hipMemcpy(dB, hB, size, hipMemcpyHostToDevice);

	//Execute the matrix multiplication kernel
	printf("\n Launching Gird of size ( %dx%d ) with Blocks of size (%d x %d)\n", GRID_SIZE, GRID_SIZE, BLOCK_SIZE, BLOCK_SIZE);
	MatrixMul <<<grid, threadBlock >>>(dA, dB, dC, N);

	// Now copy the GPU result back to CPU
	hipMemcpy(hC, dC, size, hipMemcpyDeviceToHost);

	printf("\n The Product of A and B is:\n");
	for (int j = 0; j<N; j++){
		for (int i = 0; i<N; i++){
			printf("%.2f ", hC[j*N + i]);
		}
		printf("\n");
	}
	return 0;
}


/*
output:

student@B4L0106:~$ nvcc kernel.cu
student@B4L0106:~$ ./a.out
Executing Matrix Multiplcation
Matrix size: 4 x 4<
Matrix 1:
2.00 2.00 2.00 2.00 
2.00 2.00 2.00 2.00 
2.00 2.00 2.00 2.00 
2.00 2.00 2.00 2.00 

Matrix 2:
1.00 1.00 1.00 1.00 
1.00 1.00 1.00 1.00 
1.00 1.00 1.00 1.00 
1.00 1.00 1.00 1.00 

 Launching Gird of size ( 2x2 ) with Blocks of size (2 x 2)

 Block[0][0] : Thread[0][0] : Product = 8.00

 Block[0][0] : Thread[1][0] : Product = 8.00

 Block[0][0] : Thread[0][1] : Product = 8.00

 Block[0][0] : Thread[1][1] : Product = 8.00

 Block[0][1] : Thread[0][0] : Product = 8.00

 Block[0][1] : Thread[1][0] : Product = 8.00

 Block[0][1] : Thread[0][1] : Product = 8.00

 Block[0][1] : Thread[1][1] : Product = 8.00

 Block[1][1] : Thread[0][0] : Product = 8.00

 Block[1][1] : Thread[1][0] : Product = 8.00

 Block[1][1] : Thread[0][1] : Product = 8.00

 Block[1][1] : Thread[1][1] : Product = 8.00

 Block[1][0] : Thread[0][0] : Product = 8.00

 Block[1][0] : Thread[1][0] : Product = 8.00

 Block[1][0] : Thread[0][1] : Product = 8.00

 Block[1][0] : Thread[1][1] : Product = 8.00

 The Product of A and B is:
8.00 8.00 8.00 8.00 
8.00 8.00 8.00 8.00 
8.00 8.00 8.00 8.00 
8.00 8.00 8.00 8.00 */
