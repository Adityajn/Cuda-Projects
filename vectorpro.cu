
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <stdio.h>
using namespace std;

//__global__ says that function is a kernel which will be executed by GPU by one or more simultaneous thread
// must return void, pass things by reference if you wan value
// only callable form cpu
//threadIDx.x -> within current blk id of current thread in x direction [0 - (num of thread per blk-1) ]
//blockIdx.x -> id of current blk [0 - (numof blks -1) ]
//gridDim.x -> no of blks in x direction in current grid
//blockDim.x -> no of thread in x direction in current blk
__global__ void compute(int *v1,int *v2, int *v3, int N){
	//blockIdx.x (0-2) threadIdx.x (0-99)
	if(blockIdx.x==2){
		v3[(N*blockIdx.x) + threadIdx.x] = v1[((blockIdx.x-2)*N)+threadIdx.x]*v2[((blockIdx.x-1)*N)+threadIdx.x] -
				v1[((blockIdx.x-1)*N)+threadIdx.x]*v2[((blockIdx.x-2)*N)+threadIdx.x];
	}else if(blockIdx.x==1){
		v3[(N*blockIdx.x) + threadIdx.x] = v1[((blockIdx.x+1)*N)+threadIdx.x]*v2[((blockIdx.x-1)*N)+threadIdx.x] -
				v1[(N*(blockIdx.x-1))+threadIdx.x]*v2[((blockIdx.x+1)*N)+threadIdx.x];
	}else{
		v3[(N*blockIdx.x) + threadIdx.x] = v1[((blockIdx.x+1)*N)+threadIdx.x]*v2[((blockIdx.x+2)*N)+threadIdx.x] -
				v2[((blockIdx.x+1)*N)+threadIdx.x]*v1[((blockIdx.x+2)*N)+threadIdx.x];
	}
}

// __host__ says function will be executed by host, all function without prefex are also host function
//only callable from cpu
__host__ void compute2(){    }

//__device__ says function will be executed by kernel once per call
//can be called from gpu only inside kernel or another device function
__device__ void compute3(){   }


int main(){
	int *v1,*v2,*v3;
	int *v1_d,*v2_d, *v3_d;
	v1 = (int*)malloc(sizeof(int)*300);
	v2 = (int*)malloc(sizeof(int)*300);
	v3 = (int*)malloc(sizeof(int)*300);

	int N=100;

	for(int i=0;i<N;i++){
		v1[i]=i*3;v2[i]=i*3+N;
		v1[i+N]=i*3+1;v2[i+N]=i*3+1+N;
		v1[i+2*N]=i*3+2;v2[i+2*N]=i*3+2+N;
	}
	for(int i=0;i<N;i++){
		cout<<v1[i]<<" "<<v1[i+N]<<" "<<v1[i+2*N]<<"\t"<<v2[i]<<" "<<v2[i+N]<<" "<<v2[i+2*N]<<endl;
	}

	hipMalloc((void**)&v1_d,sizeof(int)*300);
	hipMalloc((void**)&v2_d,sizeof(int)*300);
	hipMalloc((void**)&v3_d,sizeof(int)*300);
	hipMemcpy(v1_d,v1,sizeof(int)*300,hipMemcpyHostToDevice);
	hipMemcpy(v2_d,v2,sizeof(int)*300,hipMemcpyHostToDevice);


	//COMPUTE<<< NO_OF_BLKS , THREADS_PER_BLK >>>	//MAX_THREAD_PER_BLK=1024
	compute<<<3,N>>>(v1_d,v2_d,v3_d,N);	// no_of_blk=3 thread_per_blk=100

	hipMemcpy(v3,v3_d,sizeof(int)*300,hipMemcpyDeviceToHost);
	for(int i=0;i<N;i++){
		cout<<v3[i]<<"\t"<<v3[i+N]<<"\t"<<v3[i+N*2]<<endl;
	}
	cout<<"done";
	return 0;
}

/*
 WORK FLOW

 declare variable
 allocate host memory
 allocate device memory for gpu results
 write to host memory
 copy from hot to device
 execute kernel
 write gpu results in hot memory
 free host memory
 free device memory
*/
